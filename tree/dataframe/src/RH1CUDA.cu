#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <stdio.h>
#include <string>
#include <thrust/binary_search.h>

#include "RH1CUDA.h"
#include "TError.h"
#include "TMath.h"

using namespace std;

// TODO: reuse from RooBatchCompute.
#ifdef __HIPCC__
#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      Fatal((func + "(), " + file + ":" + std::to_string(line)).c_str(), "%s", hipGetErrorString(error));
      throw std::bad_alloc();
   }
}
#endif

////////////////////////////////////////////////////////////////////////////////
/// Find bin number corresponding to abscissa x. NOTE: this method does not work with alphanumeric bins !!!
///
/// If x is underflow or overflow, attempt to extend the axis if TAxis::kCanExtend is true.
/// Otherwise, return 0 or fNbins+1.

// Int_t TAxis::FindBin(Double_t x)
// {
//    Int_t bin;
//    // NOTE: This should not be allowed for Alphanumeric histograms,
//    // but it is heavily used (legacy) in the TTreePlayer to fill alphanumeric histograms.
//    // but in case of alphanumeric do-not extend the axis. It makes no sense
//    if (IsAlphanumeric() && gDebug) Info("FindBin","Numeric query on alphanumeric axis - Sorting the bins or extending
//    the axes / rebinning can alter the correspondence between the label and the bin interval."); if (x < fXmin) {
//    //*-* underflow
//       bin = 0;
//       if (fParent == 0) return bin;
//       if (!CanExtend() || IsAlphanumeric() ) return bin;
//       ((TH1*)fParent)->ExtendAxis(x,this);
//       return FindFixBin(x);
//    } else  if ( !(x < fXmax)) {     //*-* overflow  (note the way to catch NaN)
//       bin = fNbins+1;
//       if (fParent == 0) return bin;
//       if (!CanExtend() || IsAlphanumeric() ) return bin;
//       ((TH1*)fParent)->ExtendAxis(x,this);
//       return FindFixBin(x);
//    } else {
//       if (!fXbins.fN) {        //*-* fix bins
//          bin = 1 + int (fNbins*(x-fXmin)/(fXmax-fXmin) );
//       } else {                  //*-* variable bin sizes
//          //for (bin =1; x >= fXbins.fArray[bin]; bin++);
//          bin = 1 + TMath::BinarySearch(fXbins.fN,fXbins.fArray,x);
//       }
//    }
//    return bin;
// }

/// Binary search in an array of n values to locate value.
///
/// Array is supposed  to be sorted prior to this call.
/// If match is found, function returns position of element.
/// If no match found, function gives nearest element smaller than value.
#ifdef __HIPCC__
template <typename T>
__roodevice__ Long64_t BinarySearchCUDA(Long64_t n, const T  *array, T value)
{
   const T* pind;

   pind = thrust::lower_bound(thrust::seq, array, array + n, value);
   // printf("%lld %f %f %f %f\n", n, array[0], array[n], value, pind[0]);

   // if ( (pind != array + n) && (*pind == value) )
   //    return (pind - array);
   // else
   //    return ( pind - array - 1);

   return pind - array - !((pind != array + n) && (*pind == value)); // is this better?
}
#endif

__roodevice__ inline Int_t FindFixBin(Double_t x, Double_t *binEdges, Int_t nBins, Double_t xMin, Double_t xMax)
{
   Int_t bin;
   Int_t nCells = nBins - 2; // number of bins excluding U/Overflow

   // TODO: optimization -> can this be done without branching?
   if (x < xMin) { //*-* underflow
      bin = 0;
   } else if (!(x < xMax)) { //*-* overflow  (note the way to catch NaN)
      bin = nCells + 1;
   } else {
      if (binEdges == NULL) {        //*-* fix bins
         bin = 1 + int(nCells * (x - xMin) / (xMax - xMin));
      } else {                       //*-* variable bin sizes
         bin = 1 + BinarySearchCUDA(nBins - 1, binEdges, x);
      }
   }

   return bin;
}

__rooglobal__ void HistoKernel(Double_t *histogram, Double_t *binEdges, Double_t xMin, Double_t xMax, Int_t nCells,
                               Double_t *cells, Double_t *w, Size_t bufferSize, HistStats *stats)
{
   extern __shared__ Double_t block_histogram[];
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int local_tid = threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   // Initialize a local per-block histogram
   if (local_tid < nCells)
      block_histogram[local_tid] = 0;
   __syncthreads();

   // Fill local histogram
   for (int i = tid; i < bufferSize; i += stride) {
      auto bin = FindFixBin(cells[i], binEdges, nCells, xMin, xMax);
      // printf("%d: add %f to bin %d\n", tid, w[i], bin);
      atomicAdd(&block_histogram[bin], w[i]);
   }
   __syncthreads();

   // Merge results in global histogram
   if (local_tid < nCells) {
      atomicAdd(&histogram[local_tid], block_histogram[local_tid]);
   }
}

__rooglobal__ void H1DKernelGlobal(Double_t *histogram, Double_t *binEdges, Double_t xMin, Double_t xMax,
                                   Int_t nCells, Double_t *cells, Double_t *w, Size_t bufferSize)
{
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int stride = blockDim.x * gridDim.x;

   // Fill histogram
   for (int i = tid; i < bufferSize; i += stride) {
      auto bin = FindFixBin(cells[i], binEdges, nCells, xMin, xMax);
      atomicAdd(&histogram[bin], w[i]);
   }
}

// Default constructor
RHnCUDA::RHnCUDA()
{
   fThreadBlockSize = 512;
   fBufferSize = 10000;

   fNcells = 0;
   fXmin = 0;
   fXmax = 1;

   fDeviceCells = NULL;
   fDeviceWeights = NULL;
   fDeviceBinEdges = NULL;
   fBinEdges = NULL;
}

// RH1CUDA::RH1CUDA(Int_t _ncells) : RH1CUDA() {
RHnCUDA::RHnCUDA(Int_t _nCells, Double_t _xLow, Double_t _xHigh, const Double_t *_binEdges) : RHnCUDA()
{
   fNcells = _nCells;
   fXmin = _xLow;
   fXmax = _xHigh;
   fBinEdges = _binEdges;
}

// Allocate buffers for histogram on GPU
void RHnCUDA::AllocateH1D()
{
   // Allocate histogram on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceHisto, fNcells * sizeof(Double_t)));
   ERRCHECK(hipMemset(fDeviceHisto, 0, fNcells * sizeof(Double_t)));

   // Allocate weights array on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceWeights, fBufferSize * sizeof(Double_t)));

   // Allocate array of cells to fill on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceCells, fBufferSize * sizeof(Double_t)));

   if (fBinEdges != NULL) {
      ERRCHECK(hipMalloc((void **)&fDeviceBinEdges, (fNcells - 1) * sizeof(Double_t)));
      ERRCHECK(hipMemcpy(fDeviceBinEdges, fBinEdges, (fNcells - 1) * sizeof(Double_t), hipMemcpyHostToDevice));
      printf("fBinEdges\n:");
      for (int j = 0; j < fNcells - 1; ++j) {
            printf("%f ", fBinEdges[j]);
      }
      printf("\n");
   }

   ERRCHECK(hipMalloc((void **)&fDeviceStats, sizeof(HistStats)));
}

void RHnCUDA::ExecuteCUDAH1D()
{
   Size_t size = fmin(fBufferSize, fCells.size());
   // printf("cellsize:%lu buffersize:%f Size:%f nCells:%d\n", fCells.size(), fBufferSize, size, fNcells);

   ERRCHECK(hipMemcpy(fDeviceCells, fCells.data(), size * sizeof(Double_t), hipMemcpyHostToDevice));
   ERRCHECK(hipMemcpy(fDeviceWeights, fWeights.data(), size * sizeof(Double_t), hipMemcpyHostToDevice));

   HistoKernel<<<size / fThreadBlockSize + 1, fThreadBlockSize, fNcells * sizeof(Double_t)>>>(
      fDeviceHisto, fDeviceBinEdges, fXmin, fXmax, fNcells, fDeviceCells, fDeviceWeights, size, fDeviceStats);
   ERRCHECK(hipGetLastError());

   fCells.clear();
   fWeights.clear();
}

void RHnCUDA::Fill(Double_t x, Double_t w)
{
   fCells.push_back(x);
   fWeights.push_back(w);

   if (fCells.size() == fBufferSize) {
      ExecuteCUDAH1D();
   }
}

void RHnCUDA::Fill(Double_t x)
{
   Fill(x, 1.0);
}

void RHnCUDA::Fill(const char *namex, Double_t w)
{
   Fatal(":Fill(const char *namex, Double_t w)", "Cuda version not implemented yet");
}

void RHnCUDA::RetrieveResults(Double_t *result)
{
   // Fill remaning values in the histogram.
   if (fCells.size() > 0) {
      ExecuteCUDAH1D();
   }

   ERRCHECK(hipMemcpy(result, fDeviceHisto, fNcells * sizeof(Double_t), hipMemcpyDeviceToHost));
   ERRCHECK(hipMemcpy(fStats, fDeviceStats, sizeof(HistStats), hipMemcpyDeviceToHost));
   // Free device pointers?
}
