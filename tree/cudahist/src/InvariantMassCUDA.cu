#include "hip/hip_runtime.h"
#include "InvariantMassCUDA.h"
#include "Math/Vector4D.h"

#include "TError.h"

#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      Fatal((func + "(), " + file + ":" + std::to_string(line)).c_str(), "%s", hipGetErrorString(error));
      throw std::bad_alloc();
   }
}

using ROOT::Math::LorentzVector;
using ROOT::Math::PtEtaPhiE4D;

namespace ROOT {
namespace Experimental {

template <class T>
struct PtEtaPhiE4DCUDA {
   T fPt, fEta, fPhi, fE;
};

template <class T>
__global__ void
InvariantMassesKernel(const PtEtaPhiE4DCUDA<T> *v1, const PtEtaPhiE4DCUDA<T> *v2, size_t size, T *result)
{
   unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
   unsigned int stride = blockDim.x * gridDim.x;

   for (int i = tid; i < size; i += stride) {
      // Conversion from (pt, eta, phi, mass) to (x, y, z, e) coordinate system
      const auto x1 = v1[i].fPt * cos(v1[i].fPhi);
      const auto y1 = v1[i].fPt * sin(v1[i].fPhi);
      const auto z1 = v1[i].fPt * sinh(v1[i].fEta);
      const auto e1 = v1[1].fE;

      const auto x2 = v2[i].fPt * cos(v2[i].fPhi);
      const auto y2 = v2[i].fPt * sin(v2[i].fPhi);
      const auto z2 = v2[i].fPt * sinh(v2[i].fEta);
      const auto e2 = v2[i].fE;

      // Addition of particle four-vector elements
      const auto e = e1 + e2;
      const auto x = x1 + x2;
      const auto y = y1 + y2;
      const auto z = z1 + z2;

      auto mm = e * e - x * x - y * y - z * z;
      result[i] = sqrt(mm);
   }
}

template <class CoordSystem, unsigned int BlockSize>
typename CoordSystem::Scalar *
InvariantMassCUDA<CoordSystem, BlockSize>::ComputeInvariantMasses(const LorentzVector<CoordSystem> *v1, const LorentzVector<CoordSystem> *v2,
                                          size_t size)
{
   typedef typename CoordSystem::Scalar Scalar;

   const int numBlocks = ceil(size / float(BlockSize));
   printf("numblocks: %d\n", numBlocks);

   PtEtaPhiE4DCUDA<Scalar> *dV1 = NULL;
   ERRCHECK(hipMalloc((void **)&dV1, size * sizeof(PtEtaPhiE4DCUDA<Scalar>)));

   PtEtaPhiE4DCUDA<Scalar> *dV2 = NULL;
   ERRCHECK(hipMalloc((void **)&dV2, size * sizeof(PtEtaPhiE4DCUDA<Scalar>)));

   Scalar *dResult = NULL;
   ERRCHECK(hipMalloc((void **)&dResult, size * sizeof(Scalar)));

   ERRCHECK(hipMemcpy(dV1, v1, size * sizeof(PtEtaPhiE4DCUDA<Scalar>), hipMemcpyHostToDevice));
   ERRCHECK(hipMemcpy(dV2, v2, size * sizeof(PtEtaPhiE4DCUDA<Scalar>), hipMemcpyHostToDevice));

   InvariantMassesKernel<<<numBlocks, BlockSize>>>(dV1, dV2, size, dResult);
   hipDeviceSynchronize();
   ERRCHECK(hipPeekAtLastError());

   Scalar *result = (Scalar *)malloc(size * sizeof(Scalar));
   ERRCHECK(hipMemcpy(result, dResult, size * sizeof(Scalar), hipMemcpyDeviceToHost));
   return result;
}

// Template instantations
template class InvariantMassCUDA<PtEtaPhiE4D<double>, 256>;

} // namespace Experimental
} // namespace ROOT