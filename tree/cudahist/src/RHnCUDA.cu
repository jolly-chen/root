#include "hip/hip_runtime.h"
#include "RHnCUDA.h"

#include "CUDAHelpers.cuh"
#include "RtypesCore.h"
#include "TError.h"
#include "TMath.h"

#include <thrust/functional.h>
#include <array>
#include <vector>
#include <utility>
#include <iostream>

namespace CUDAhist {

////////////////////////////////////////////////////////////////////////////////
/// CUDA Histogram Kernels

__device__ inline int FindFixBin(double x, const double *binEdges, int nBins, double xMin, double xMax)
{
   int bin;

   // OPTIMIZATION: can this be done with less branching?
   if (x < xMin) { //*-* underflow
      bin = 0;
   } else if (!(x < xMax)) { //*-* overflow  (note the way to catch NaN)
      bin = nBins + 1;
   } else {
      if (binEdges == NULL) { //*-* fix bins
         bin = 1 + int(nBins * (x - xMin) / (xMax - xMin));
      } else { //*-* variable bin sizes
         bin = 1 + CUDAHelpers::BinarySearchCUDA(nBins + 1, binEdges, x);
      }
   }

   return bin;
}

// Use Horner's method to calculate the bin in an n-Dimensional array.
template <unsigned int Dim>
__device__ inline int GetBin(int i, CUDAhist::RAxis *axes, double *coords, int *bins)
{
   auto *x = &coords[i * Dim];

   auto bin = 0;
   for (int d = Dim - 1; d >= 0; d--) {
      auto binD = FindFixBin(x[d], axes[d].kBinEdges, axes[d].fNbins - 2, axes[d].fMin, axes[d].fMax);
      bins[i * Dim + d] = binD;

      if (binD < 0) {
         return -1;
      }

      // printf("Dim:%d  bin:%d x:%f ncells:%d min:%f max:%f\n", d, bin, x[d], axes[d].fNbins, axes[d].fMin,
      // axes[d].fMax);
      bin = bin * axes[d].fNbins + binD;
   }

   return bin;
}

template <typename T, unsigned int Dim>
__global__ void HistoKernel(T *histogram, CUDAhist::RAxis *axes, int nBins, double *coords, int *bins, double *weights,
                            unsigned int bufferSize)
{
   auto smem = CUDAHelpers::shared_memory_proxy<T>();
   unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
   unsigned int local_tid = threadIdx.x;
   unsigned int stride = blockDim.x * gridDim.x;

   // Initialize a local per-block histogram
   for (auto i = local_tid; i < nBins; i += blockDim.x) {
      smem[local_tid] = 0;
   }
   __syncthreads();

   // Fill local histogram
   for (auto i = tid; i < bufferSize; i += stride) {
      auto bin = GetBin<Dim>(i, axes, coords, bins);
      // printf("%d: add %f to bin %d\n", tid, weights[i], bin);

      // TODO: check for datatype under/overflow
      if (bin >= 0)
         atomicAdd(&smem[bin], (T)weights[i]);
   }
   __syncthreads();

   // Merge results in global histogram
   for (auto i = local_tid; i < nBins; i += blockDim.x) {
      // printf("%d: merge %f into bin %d\n", tid, smem[i], i);
      atomicAdd(&histogram[i], smem[i]);
   }
}

// Slower histogramming, but requires less memory.
// OPTIMIZATION: consider sorting the coords array.
template <typename T, unsigned int Dim>
__global__ void HistoKernelGlobal(T *histogram, CUDAhist::RAxis *axes, int nBins, double *coords, int *bins,
                                  double *weights, unsigned int bufferSize)
{
   unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
   unsigned int stride = blockDim.x * gridDim.x;

   // Fill histogram
   for (auto i = tid; i < bufferSize; i += stride) {
      auto bin = GetBin<Dim>(i, axes, coords, bins);
      if (bin >= 0)
         atomicAdd(&histogram[bin], (T)weights[i]);
   }
}

// TODO: worked for 1 Dimension, need to adapt to n-Dimensional case.
// OPTIMIZATION: interleave/change order of computation of different stats to improve coalescing?
template <unsigned int Dim, unsigned int BlockSize>
__global__ void GetStatsKernel(double *coords, int *bins, double *weights, unsigned int nCoords, CUDAhist::RAxis *axes,
                               double *fDIntermediateStats, const int nStats)
{
   unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
   unsigned int stride = blockDim.x * gridDim.x;
   auto sdata = CUDAHelpers::shared_memory_proxy<double>();

   // if (tid == 0 && Dim == 2)
   //    printf("ncoords:%i nbinsX:%i nbinsY:%d\n", nCoords, axes[0].fNbins, axes[1].fNbins);

   // Exclude under/overflow bins from stats
   for (auto i = tid; i < nCoords; i += stride) {
      for (auto d = 0; d < Dim; d++) {
         if (bins[i * Dim + d] <= 0 || bins[i * Dim + d] >= axes[d].fNbins - 1) {
            // if (Dim == 2)
            // printf("cuda SKIP? bin:%d val:%f i:%d\n", bins[i * Dim + d], coords[i * Dim + d], i * Dim + d);

            weights[i] = 0.;
            // continue;
         }
      }
      // printf("%d\n", i);
   }

   // Tsumw
   CUDAHelpers::ReduceBase<BlockSize>(
      sdata, weights, fDIntermediateStats, nCoords, [](unsigned int i, double r, double w) { return r + w; },
      CUDAHelpers::Plus<double>());

   if (tid == 0) {
      printf("sdara1:\n");
      for (int i = 0; i <  BlockSize * nStats / 2; i++) {
         if (i % ((BlockSize <= 32) ? 2 * BlockSize * nStats : BlockSize * nStats)  == 0)
            printf("\n\n");
         printf("%i%f \n", sdata[i]);
      }
      printf("\n\n");
   }

   // Tsumw2
   unsigned int sdata_offset = blockDim.x;
   unsigned int is_offset = gridDim.x;
   CUDAHelpers::ReduceBase<BlockSize>(
      &sdata[sdata_offset], weights, &fDIntermediateStats[is_offset], nCoords,
      [](unsigned int i, double r, double w) { return r + w * w; }, CUDAHelpers::Plus<double>());

   for (auto d = 0; d < Dim; d++) {
      // Multiply weight with coordinate of current axis. E.g., for Dim = 2 this computes Tsumwx and Tsumwy
      sdata_offset += blockDim.x;
      is_offset += gridDim.x;
      CUDAHelpers::ReduceBase<BlockSize>(
         &sdata[sdata_offset], weights, &fDIntermediateStats[is_offset], nCoords,
         [&coords, &d](unsigned int i, double r, double w) { return r + w * coords[i * Dim + d]; },
         CUDAHelpers::Plus<double>());

      // Squares coodinate per axis. E.g., for Dim = 2 this computes Tsumw2 and Tsumwy2
      sdata_offset += blockDim.x;
      is_offset += gridDim.x;
      CUDAHelpers::ReduceBase<BlockSize>(
         &sdata[sdata_offset], weights, &fDIntermediateStats[is_offset], nCoords,
         [&coords, &d](unsigned int i, double r, double w) {
            return r + w * coords[i * Dim + d] * coords[i * Dim + d];
         },
         CUDAHelpers::Plus<double>());

      for (auto prev_d = d - 1; prev_d >= 0; prev_d--) {
         // Multiplies coordinate of current axis with the "previous" axis. E.g., for Dim = 2 this computes Tsumwxy
         sdata_offset += blockDim.x;
         is_offset += gridDim.x;
         CUDAHelpers::ReduceBase<BlockSize>(
            &sdata[sdata_offset], weights, &fDIntermediateStats[is_offset], nCoords,
            [&coords, &prev_d, &d](unsigned int i, double r, double w) {
               return r + w * coords[i * Dim + prev_d] * coords[i * Dim + d];
            },
            CUDAHelpers::Plus<double>());
      }
   }
}

template <typename T, unsigned int Dim, unsigned int BlockSize>
RHnCUDA<T, Dim, BlockSize>::RHnCUDA(int *ncells, double *xlow, double *xhigh, const double **binEdges)
   : kNStats([]() {
        // Sum of weights (squared) + sum of weight * bin (squared) per axis + sum of weight * binAx1 * binAx2 for
        // all axis combinations
        return Dim > 1 ? 2 + 2 * Dim + TMath::Binomial(Dim, 2) : 2 + 2 * Dim;
     }()),
     kStatsSmemSize((BlockSize <= 32) ? 2 * BlockSize * kNStats * sizeof(double) : BlockSize * kNStats * sizeof(double))
// template <typename T, unsigned int Dim, unsigned int BlockSize>
// RHnCUDA<T, Dim, BlockSize>::Initialize(int *ncells, double *xlow, double *xhigh, const double **binEdges)
// RHnCUDA::Initialize(int *ncells, double *xlow, double *xhigh, const double **binEdges)
{
   fBufferSize = 10000;

   fNbins = 1;
   fEntries = 0;
   fDIntermediateStats = NULL;
   fDStats = NULL;
   fDAxes = NULL;
   fHCoords.reserve(Dim * fBufferSize);
   fHWeights.reserve(fBufferSize);

   // Initialize axis descriptors.
   for (int i = 0; i < Dim; i++) {
      RAxis axis;
      axis.fNbins = ncells[i];
      axis.fMin = xlow[i];
      axis.fMax = xhigh[i];
      axis.kBinEdges = binEdges[i];
      fHAxes[i] = axis;

      fNbins *= ncells[i];
      if (getenv("DBG"))
         printf("\t axis %d -- ncells:%d min:%f max:%f\n", i, axis.fNbins, axis.fMin, axis.fMax);
   }

   hipDeviceProp_t prop;
   ERRCHECK(hipGetDeviceProperties(&prop, 0));
   fMaxSmemSize = prop.sharedMemPerBlock;
   fHistoSmemSize = fNbins * sizeof(T);

   if (getenv("DBG"))
      printf("nbins:%d Dim:%d nstats:%d maxsmem:%d\n", fNbins, Dim, kNStats, fMaxSmemSize);
}

template <typename T, unsigned int Dim, unsigned int BlockSize>
void RHnCUDA<T, Dim, BlockSize>::AllocateH1D()
{
   // Allocate histogram on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceHisto, fNbins * sizeof(double)));
   ERRCHECK(hipMemset(fDeviceHisto, 0, fNbins * sizeof(double)));

   // Allocate weights array on GPU
   ERRCHECK(hipMalloc((void **)&fDWeights, fBufferSize * sizeof(double)));

   // Allocate array of coords to fill on GPU
   ERRCHECK(hipMalloc((void **)&fDCoords, Dim * fBufferSize * sizeof(double)));

   // Allocate array of bins corresponding to the coords.
   ERRCHECK(hipMalloc((void **)&fDBins, Dim * fBufferSize * sizeof(int)));

   // Allocate axes on the GPU
   ERRCHECK(hipMalloc((void **)&fDAxes, fBufferSize * sizeof(RAxis)));
   ERRCHECK(hipMemcpy(fDAxes, fHAxes.data(), Dim * sizeof(RAxis), hipMemcpyHostToDevice));
   for (auto i = 0; i < Dim; i++) {
      // Allocate memory for BinEdges array.
      if (fHAxes[i].kBinEdges != NULL) {
         double *deviceBinEdges;
         ERRCHECK(hipMalloc((void **)&deviceBinEdges, fHAxes[i].fNbins * sizeof(double)));
         ERRCHECK(
            hipMemcpy(deviceBinEdges, fHAxes[i].kBinEdges, fHAxes[i].fNbins * sizeof(double), hipMemcpyHostToDevice));
         ERRCHECK(hipMemcpy(&fDAxes[i].kBinEdges, &deviceBinEdges, sizeof(double *), hipMemcpyHostToDevice));
      }
   }

   // Allocate array with (intermediate) results of the stats for each block.
   ERRCHECK(hipMalloc((void **)&fDStats, kNStats * sizeof(double)));
}

// TODO: ref to array
template <typename T, unsigned int Dim, unsigned int BlockSize>
void RHnCUDA<T, Dim, BlockSize>::Fill(const std::array<T, Dim> &coords, double w)
{
   fHCoords.insert(fHCoords.end(), coords.begin(), coords.end());
   fHWeights.push_back(w);

   // Only execute when a certain number of values are buffered to increase the GPU workload and decrease the
   // frequency of kernel launches.
   if (fHWeights.size() == fBufferSize) {
      ExecuteCUDAHisto();
   }
}

template <typename T, unsigned int Dim, unsigned int BlockSize>
void RHnCUDA<T, Dim, BlockSize>::GetStats(unsigned int size)
{
   // TODO: move this to the constructor
   // Number of blocks in grid is halved, because each thread loads two elements from global memory.
   int numBlocks = fmax(1, ceil(size / BlockSize / 2.));

   if (getenv("DBG") && atoi(getenv("DBG")) > 0)
      printf("STATS -- size:%d smemsize: %lu numblocks: %d blocksize %d\n", size, kStatsSmemSize / sizeof(double),
             numBlocks, BlockSize);

   if (fDIntermediateStats == NULL)
      ERRCHECK(hipMalloc((void **)&fDIntermediateStats, numBlocks * kNStats * sizeof(double)));
   else
      ERRCHECK(hipMemset(fDIntermediateStats, 0, numBlocks * kNStats * sizeof(double)));

   GetStatsKernel<Dim, BlockSize><<<numBlocks, BlockSize, kStatsSmemSize>>>(fDCoords, fDBins, fDWeights, size, fDAxes,
                                                                            fDIntermediateStats, kNStats);
   ERRCHECK(hipPeekAtLastError());

   // OPTIMIZATION: final reduction in a single kernel?
   for (auto i = 0; i < kNStats; i++) {
      CUDAHelpers::ReductionKernel<BlockSize, double, false><<<1, BlockSize, kStatsSmemSize>>>(
         &fDIntermediateStats[i * numBlocks], &fDStats[i], numBlocks, CUDAHelpers::Plus<double>(), 0.);
      ERRCHECK(hipPeekAtLastError());
   }
}

template <typename T, unsigned int Dim, unsigned int BlockSize>
void RHnCUDA<T, Dim, BlockSize>::ExecuteCUDAHisto()
{
   unsigned int size = fmin(fBufferSize, fHWeights.size());
   int numBlocks = size % BlockSize == 0 ? size / BlockSize : size / BlockSize + 1;

   if (getenv("DBG") && atoi(getenv("DBG")) > 2) {
      printf("HISTO -- cellsize:%lu buffersize:%d Size:%d nCells:%d nBlocks:%d smemsize:%u\n", fHCoords.size(),
             fBufferSize, size, fNbins, numBlocks, fHistoSmemSize);
   }

   fEntries += size;

   ERRCHECK(hipMemcpy(fDCoords, fHCoords.data(), Dim * size * sizeof(double), hipMemcpyHostToDevice));
   ERRCHECK(hipMemcpy(fDWeights, fHWeights.data(), size * sizeof(double), hipMemcpyHostToDevice));

   if (fHistoSmemSize > fMaxSmemSize) {
      HistoKernelGlobal<T, Dim>
         <<<numBlocks, BlockSize>>>(fDeviceHisto, fDAxes, fNbins, fDCoords, fDBins, fDWeights, size);
   } else {
      HistoKernel<T, Dim>
         <<<numBlocks, BlockSize, fHistoSmemSize>>>(fDeviceHisto, fDAxes, fNbins, fDCoords, fDBins, fDWeights, size);
   }
   ERRCHECK(hipPeekAtLastError());

   GetStats(size);

   fHCoords.clear();
   fHWeights.clear();
}

template <typename T, unsigned int Dim, unsigned int BlockSize>
int RHnCUDA<T, Dim, BlockSize>::RetrieveResults(double *histResult, double *statsResult)
{
   // Fill the histogram with remaining values in the buffer.
   if (fHWeights.size() > 0) {
      ExecuteCUDAHisto();
   }

   // Copy back results from GPU to CPU.
   ERRCHECK(hipMemcpy(histResult, fDeviceHisto, fNbins * sizeof(double), hipMemcpyDeviceToHost));
   ERRCHECK(hipMemcpy(statsResult, fDStats, kNStats * sizeof(double), hipMemcpyDeviceToHost));

   // TODO: Free device pointers?

   return fEntries;
}

#include "RHnCUDA-impl.cu"

} // namespace CUDAhist
