#include "hip/hip_runtime.h"
#include <vector>
#include <stdio.h>
#include <thrust/functional.h>

#include "RHnCUDA.h"
#include "CUDAHelpers.cuh"
#include "TMath.h"

using namespace std;
// using namespace erge;

////////////////////////////////////////////////////////////////////////////////
/// CUDA Histogram Kernels

__device__ inline Int_t FindFixBin(Double_t x, const Double_t *binEdges, Int_t nBins, Double_t xMin, Double_t xMax)
{
   Int_t bin;

   // TODO: optimization -> can this be done without branching?
   if (x < xMin) { //*-* underflow
      bin = 0;
   } else if (!(x < xMax)) { //*-* overflow  (note the way to catch NaN)
      bin = nBins + 1;
   } else {
      if (binEdges == NULL) { //*-* fix bins
         bin = 1 + int(nBins * (x - xMin) / (xMax - xMin));
      } else { //*-* variable bin sizes
         bin = 1 + CUDAHelpers::BinarySearchCUDA(nBins + 1, binEdges, x);
      }
   }

   return bin;
}

// Use Horner's method to calculate the bin in an n-dimensional array.
__device__ inline Int_t GetBin(Int_t i, Int_t dim, RHnCUDA::RAxis *axes, Double_t *cells)
{
   auto *x = &cells[i * dim];

   auto d = dim-1;
   auto bin = FindFixBin(x[d], axes[d].kBinEdges, axes[d].fNcells - 2, axes[d].fMin, axes[d].fMax);
   // printf("dim:%d  bin:%d x:%f ncells:%d min:%f max:%f\n", d, bin, x[d], axes[d].fNcells, axes[d].fMin, axes[d].fMax);

   for (d--; d >= 0; d--) {
      auto binD = FindFixBin(x[d], axes[d].kBinEdges, axes[d].fNcells - 2, axes[d].fMin, axes[d].fMax);
      if (binD < 0) return -1;
      // printf("dim:%d  bin:%d x:%f ncells:%d min:%f max:%f\n", d, bin, x[d], axes[d].fNcells, axes[d].fMin, axes[d].fMax);
      bin = bin * axes[d].fNcells + binD;
   }

   return bin;
}

__global__ void HistoKernel(Double_t *histogram, Int_t dim, RHnCUDA::RAxis *axes, Int_t nBins, Double_t *cells,
                            Double_t *w, UInt_t bufferSize, RHnCUDA::Stats *stats)
{
   extern __shared__ Double_t smem[];
   int tid = threadIdx.x + blockDim.x * blockIdx.x;
   int local_tid = threadIdx.x;
   int stride = blockDim.x * gridDim.x;

   // Initialize a local per-block histogram
   if (local_tid < nBins)
      smem[local_tid] = 0;
   __syncthreads();

   // Fill local histogram
   for (int i = tid; i < bufferSize; i += stride) {
      auto bin = GetBin(i, dim, axes, cells);
      // printf("%d: add %f to bin %d\n", tid, w[i], bin);
      if (bin < 0) continue;
      atomicAdd(&smem[bin], w[i]);

      // Don't include u/overflow bins in stats.
      // TODO: maybe not very clean to be modifying the input weights array.
      if (bin == 0 || bin > nBins)
         w[i] = 0;
   }
   __syncthreads();

   // Merge results in global histogram
   if (local_tid < nBins) {
      atomicAdd(&histogram[local_tid], smem[local_tid]);
   }
}

// template <UInt_t BlockSize, typename ValType>
// __global__ void GetStatsKernel(ValType *cells, Double_t *weights, Double_t *oSumw, Double_t *oSumw2, Double_t
// *oSumwx,
//                                Double_t *oSumwx2, UInt_t n)
// {
//    extern __shared__ Double_t sdata[];

//    UInt_t tid = threadIdx.x;
//    UInt_t i = blockIdx.x * (BlockSize * 2) + tid;
//    UInt_t gridSize = (BlockSize * 2) * gridDim.x;

//    // Only one shared memory buffer can be declared so we index with an offset to differentiate multiple arrays.
//    Double_t *sdataSumw = &sdata[0];
//    Double_t *sdataSumw2 = &sdata[blockDim.x];
//    Double_t *sdataSumwx = &sdata[2 * blockDim.x];
//    Double_t *sdataSumwx2 = &sdata[3 * blockDim.x];

//    // if (i == 0) {
//    //    printf("blockdim:%d griddim:%d gridsize:%d\n", blockDim.x, gridDim.x, gridSize);
//    // }

//    // Operate on local var instead of sdata to avoid illegal memory accesses?
//    Double_t rsumw = 0, rsumw2 = 0, rsumwx = 0, rsumwx2 = 0.;

//    while (i < n) {
//       rsumw += weights[i];
//       rsumw2 = weights[i] * weights[i];
//       rsumwx = weights[i] * cells[i];
//       rsumwx2 = weights[i] * cells[i] * cells[i];

//       if (i + BlockSize < n) {
//          rsumw += weights[i + BlockSize];
//          rsumw2 += weights[i + BlockSize] * weights[i + BlockSize];
//          rsumwx += weights[i + BlockSize] * cells[i + BlockSize];
//          rsumwx2 += weights[i + BlockSize] * cells[i + BlockSize] * cells[i + BlockSize];
//       }

//       i += gridSize;
//    }
//    sdataSumw[tid] = rsumw;
//    sdataSumw2[tid] = rsumw2;
//    sdataSumwx[tid] = rsumwx;
//    sdataSumwx2[tid] = rsumwx2;
//    __syncthreads();

//    CUDAHelpers::UnrolledReduce<BlockSize, CUDAHelpers::plus<Double_t>, Double_t>(sdataSumw, tid);
//    CUDAHelpers::UnrolledReduce<BlockSize, CUDAHelpers::plus<Double_t>, Double_t>(sdataSumw2, tid);
//    CUDAHelpers::UnrolledReduce<BlockSize, CUDAHelpers::plus<Double_t>, Double_t>(sdataSumwx, tid);
//    CUDAHelpers::UnrolledReduce<BlockSize, CUDAHelpers::plus<Double_t>, Double_t>(sdataSumwx2, tid);

//    // The first thread of each block writes the sum of the block into the global device array.
//    if (tid == 0) {
//       oSumw[blockIdx.x] = sdataSumw[0];
//       oSumw2[blockIdx.x] = sdataSumw2[0];
//       oSumwx[blockIdx.x] = sdataSumwx[0];
//       oSumwx2[blockIdx.x] = sdataSumwx2[0];
//    }
// }

// template __global__ void GetStatsKernel<512, Double_t>(Double_t *cells, Double_t *weights, Double_t *oSumw,
//                                                        Double_t *oSumw2, Double_t *oSumwx, Double_t *oSumwx2, UInt_t
//                                                        n);

// __global__ void H1DKernelGlobal(Double_t *histogram, Double_t *binEdges, Double_t xMin, Double_t xMax, Int_t nCells,
//                                 Double_t *cells, Double_t *w, UInt_t bufferSize)
// {
//    int tid = threadIdx.x + blockDim.x * blockIdx.x;
//    int stride = blockDim.x * gridDim.x;

//    // Fill histogram
//    for (int i = tid; i < bufferSize; i += stride) {
//       auto bin = FindFixBin(cells[i], binEdges, nCells, xMin, xMax);
//       // printf("%d: add %f to bin %d\n", tid, w[i], bin);
//       atomicAdd(&histogram[bin], w[i]);
//    }
// }

////////////////////////////////////////////////////////////////////////////////
/// RHnCUDA constructor

RHnCUDA::RHnCUDA(Int_t dim, Int_t *ncells, Double_t *xlow, Double_t *xhigh, const Double_t **binEdges) : kDim(dim)
{
   fThreadBlockSize = 512;
   fBufferSize = 10000;

   fNbins = 1;
   fEntries = 0;
   fDeviceStats = NULL;
   fDeviceAxes = NULL;
   fCells.reserve(dim * fBufferSize);
   fWeights.reserve(fBufferSize);

   for (int i = 0; i < dim; i++) {
      RAxis axis;
      axis.fNcells = ncells[i];
      axis.fMin = xlow[i];
      axis.fMax = xhigh[i];
      axis.kBinEdges = binEdges[i];
      fAxes.push_back(axis);

      fNbins *= ncells[i];
      if (getenv("DBG")) printf("ncells:%d min:%f max:%f ", axis.fNcells, axis.fMin, axis.fMax);
   }
   if (getenv("DBG")) printf("nbins:%d dim:%d\n", fNbins, kDim);
}

// Allocate buffers for histogram on GPU
void RHnCUDA::AllocateH1D()
{
   // Allocate histogram on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceHisto, fNbins * sizeof(Double_t)));
   ERRCHECK(hipMemset(fDeviceHisto, 0, fNbins * sizeof(Double_t)));

   // Allocate weights array on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceWeights, fBufferSize * sizeof(Double_t)));

   // Allocate array of cells to fill on GPU
   ERRCHECK(hipMalloc((void **)&fDeviceCells, kDim * fBufferSize * sizeof(Double_t)));

   // Allocate axes on the GPU
   ERRCHECK(hipMalloc((void **)&fDeviceAxes, fBufferSize * sizeof(RAxis)));
   ERRCHECK(hipMemcpy(fDeviceAxes, fAxes.data(), kDim * sizeof(RAxis), hipMemcpyHostToDevice));
   for (int i = 0; i < kDim; i++) {
      // Allocate memory for BinEdges array.
      if (fAxes[i].kBinEdges != NULL) {
         // ERRCHECK(hipMalloc((void **)fDeviceAxes[i].fBinEdges, fAxes[i].fNcells * sizeof(Double_t)));
         // ERRCHECK(hipMemcpy(&fDeviceAxes[i].fBinEdges, fAxes[i].fBinEdges, fAxes[i].fNcells * sizeof(Double_t),
         //                     hipMemcpyHostToDevice));

         Double_t *deviceBinEdges;
         ERRCHECK(hipMalloc((void **)&deviceBinEdges, fAxes[i].fNcells * sizeof(Double_t)));
         ERRCHECK(hipMemcpy(deviceBinEdges, fAxes[i].kBinEdges, fAxes[i].fNcells * sizeof(Double_t),
                             hipMemcpyHostToDevice));
         ERRCHECK(hipMemcpy(&fDeviceAxes[i].kBinEdges, &deviceBinEdges, sizeof(Double_t *), hipMemcpyHostToDevice));
      }
   }

   ERRCHECK(hipMalloc((void **)&fDeviceStats, sizeof(RHnCUDA::Stats)));
   ERRCHECK(hipMemset(fDeviceStats, 0, 5 * sizeof(Double_t))); // set the first 5 variables in the struct to 0.
}

void RHnCUDA::GetStats(UInt_t size)
{
   // const UInt_t blockSize = 512;

   // Int_t smemSize = (blockSize <= 32) ? 2 * blockSize : blockSize;
   // UInt_t numBlocks = fmax(1, ceil(size / blockSize / 2.)); // Number of blocks in grid is halved!

   // Double_t *intermediate_sumw = NULL;
   // Double_t *intermediate_sumw2 = NULL;
   // Double_t *intermediate_sumwx = NULL;
   // Double_t *intermediate_sumwx2 = NULL;
   // ERRCHECK(hipMalloc((void **)&intermediate_sumw, numBlocks * sizeof(Double_t)));
   // ERRCHECK(hipMalloc((void **)&intermediate_sumw2, numBlocks * sizeof(Double_t)));
   // ERRCHECK(hipMalloc((void **)&intermediate_sumwx, numBlocks * sizeof(Double_t)));
   // ERRCHECK(hipMalloc((void **)&intermediate_sumwx2, numBlocks * sizeof(Double_t)));

   // GetStatsKernel<blockSize, Double_t><<<numBlocks, blockSize, 4 * smemSize * sizeof(Double_t)>>>(
   //    fDeviceCells, fDeviceWeights, intermediate_sumw, intermediate_sumw2, intermediate_sumwx, intermediate_sumwx2,
   //    size);
   // ERRCHECK(hipGetLastError());
   // // OPTIMIZATION: final reduction in a single kernel?
   // CUDAHelpers::ReductionKernel<blockSize, CUDAHelpers::plus<Double_t>, Double_t, false>
   //    <<<1, blockSize, smemSize * sizeof(Double_t)>>>(intermediate_sumw, &(fDeviceStats->fTsumw), numBlocks, 0.);
   // ERRCHECK(hipGetLastError());
   // CUDAHelpers::ReductionKernel<blockSize, CUDAHelpers::plus<Double_t>, Double_t, false>
   //    <<<1, blockSize, smemSize * sizeof(Double_t)>>>(intermediate_sumw2, &(fDeviceStats->fTsumw2), numBlocks, 0.);
   // ERRCHECK(hipGetLastError());
   // CUDAHelpers::ReductionKernel<blockSize, CUDAHelpers::plus<Double_t>, Double_t, false>
   //    <<<1, blockSize, smemSize * sizeof(Double_t)>>>(intermediate_sumwx, &(fDeviceStats->fTsumwx), numBlocks, 0.);
   // ERRCHECK(hipGetLastError());
   // CUDAHelpers::ReductionKernel<blockSize, CUDAHelpers::plus<Double_t>, Double_t, false>
   //    <<<1, blockSize, smemSize * sizeof(Double_t)>>>(intermediate_sumwx2, &(fDeviceStats->fTsumwx2), numBlocks, 0.);
   // ERRCHECK(hipGetLastError());

   // ERRCHECK(hipFree(intermediate_sumw));
   // ERRCHECK(hipFree(intermediate_sumw2));
   // ERRCHECK(hipFree(intermediate_sumwx));
   // ERRCHECK(hipFree(intermediate_sumwx2));
}

void RHnCUDA::ExecuteCUDAH1D()
{
   UInt_t size = fmin(fBufferSize, fWeights.size());
   // printf("cellsize:%lu buffersize:%f Size:%f nCells:%d\n", fCells.size(), fBufferSize, size, fNcells);

   fEntries += size;

   ERRCHECK(hipMemcpy(fDeviceCells, fCells.data(), kDim * size * sizeof(Double_t), hipMemcpyHostToDevice));
   ERRCHECK(hipMemcpy(fDeviceWeights, fWeights.data(), size * sizeof(Double_t), hipMemcpyHostToDevice));

   // TODO: this fails with invalid argument when  fNbins * sizeof(Double_t) exceeds max shared mem size.
   HistoKernel<<<size / fThreadBlockSize + 1, fThreadBlockSize, fNbins * sizeof(Double_t)>>>(
      fDeviceHisto, kDim, fDeviceAxes, fNbins, fDeviceCells, fDeviceWeights, size, fDeviceStats);
   ERRCHECK(hipGetLastError());
   GetStats(size);

   fCells.clear();
   fWeights.clear();
}

void RHnCUDA::Fill(std::vector<Double_t> x, Double_t w)
{
   if (x.size() != kDim)
      return;

   fCells.insert(fCells.end(), x.begin(), x.end());
   fWeights.push_back(w);

   // Only execute when a certain number of values are buffered to increase the GPU workload and decrease the
   // frequency of kernel launches.
   if (fWeights.size() == fBufferSize) {
      ExecuteCUDAH1D();
   }
}

void RHnCUDA::Fill(std::vector<Double_t> x)
{
   Fill(x, 1.0);
}

void RHnCUDA::Fill(const char *namex, Double_t w)
{
   Fatal(":Fill(const char *namex, Double_t w)", "Cuda version not implemented yet");
}

// Copy back results on GPU to CPU.
Int_t RHnCUDA::RetrieveResults(Double_t *histResult, Double_t *statsResult)
{
   // Fill the histogram with remaining values in the buffer.
   if (fCells.size() > 0) {
      ExecuteCUDAH1D();
   }

   Stats stats;
   ERRCHECK(hipMemcpy(histResult, fDeviceHisto, fNbins * sizeof(Double_t), hipMemcpyDeviceToHost));
   ERRCHECK(hipMemcpy(&stats, fDeviceStats, sizeof(Stats), hipMemcpyDeviceToHost));
   statsResult[0] = stats.fTsumw;
   statsResult[1] = stats.fTsumw2;
   statsResult[2] = stats.fTsumwx;
   statsResult[3] = stats.fTsumwx2;

   // // TODO: Free device pointers?

   return fEntries;
}
